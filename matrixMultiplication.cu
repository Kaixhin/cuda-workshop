#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#define THREADS_PER_BLOCK 1024

void matrixMultiplyCPU(float *a, float *b, float *c, int width) {
  float result;

  for (int row = 0; row < width; row++) {
    for (int col = 0; col < width; col++) {
      result = 0;
      for (int k = 0; k < width; k++) {
        result += a[row * width + k] * b[k * width + col];
      }
      c[row * width + col] = result;
    }
  }
}

__global__ void matrixMultiplySimple(float *a, float *b, float *c, int width) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  float result = 0;

  if (col < width && row < width) {
    for (int k = 0; k < width; k++) {
      result += a[row * width + k] * b[k * width + col];
    }
    c[row * width + col] = result;
  }
}

const int TILE_WIDTH = 16;
__global__ void matrixMultiplyOptimised(float *a, float *b, float *c, int width) {
  // Allocate 2D tiles in shared memory
  __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
  __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

  // Calculate row and column index of element
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float result = 0;
  
  // Loop over tiles of input in phases
  for (int p = 0; p < width/TILE_WIDTH; p++) {
    // Collaboratively load tiles into shared memory
    s_a[threadIdx.y][threadIdx.x] = a[row*width + (p*TILE_WIDTH + threadIdx.x)];
    s_b[threadIdx.y][threadIdx.x] = b[(p*TILE_WIDTH + threadIdx.y)*width + col];

    // Wait until all data is loaded before allowing any threads in the block to continue
    __syncthreads();

    // Dot product between row of s_a and column of s_b
    for (int i = 0; i < TILE_WIDTH; i++) {
      result += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
    }

    // Wait until all calculations are finished before allowing any threads in the block to continue
    __syncthreads();
  }

  // Write result
  c[row * width + col] = result;
}

int main() {
  int width = 2000; // Define width of square matrix
  // Initialise grid and block variables
  int sqrtThreads = sqrt(THREADS_PER_BLOCK);
  int nBlocks = width/sqrtThreads;
  if (width % sqrtThreads != 0) { // Add an extra block if necessary
    nBlocks++;
  }
  dim3 grid(nBlocks, nBlocks, 1);
  dim3 block(sqrtThreads, sqrtThreads, 1); // Max number of threads per block

  // Initialise host pointers (dynamically allocated memory) and device pointers
  float *a_h;
  float *b_h;
  float *c_h; // GPU results
  float *d_h; // CPU results
  float *a_d;
  float *b_d;
  float *c_d;

  int size; // Number of bytes required by arrays

  // Create timer
  hipEvent_t start;
  hipEvent_t stop;
  float elapsed1, elapsed2, elapsed3;

  // Print out information about blocks and threads
  printf("Number of threads: %i (%ix%i)\n", block.x*block.y, block.x, block.y);
  printf("Number of blocks: %i (%ix%i)\n", grid.x*grid.y, grid.x, grid.y);

  // Dynamically allocate host memory
  size = width * width * sizeof(float);
  
  a_h = (float*) malloc(size);
  b_h = (float*) malloc(size);
  c_h = (float*) malloc(size);
  d_h = (float*) malloc(size);

  // Load host arrays with data
  for (int i = 0; i < width; i++) {
    for (int j = 0; j < width; j++) {
      a_h[i * width + j] = i;
      b_h[i * width + j] = i;
    }
  }

  // Allocate device memory
  hipMalloc((void**)&a_d, size);
  hipMalloc((void**)&b_d, size);
  hipMalloc((void**)&c_d, size);

  // Copy host memory to device memory
  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
  hipMemcpy(c_d, c_h, size, hipMemcpyHostToDevice);

  // Start timer for GPU
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Launch kernel
  matrixMultiplySimple<<<grid, block>>>(a_d, b_d, c_d, width);

  // Stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed1, start, stop);

  // Print execution time
  printf("Time to calculate results on GPU: %f ms\n", elapsed1);

  // Copy results to host
  hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
  
  // Start timer for CPU
  hipEventRecord(start, 0);

  // Launch CPU code
  matrixMultiplyCPU(a_h, b_h, d_h, width);

  // Stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed2, start, stop);

  // Print execution time
  printf("Time to calculate results on CPU: %f ms\n", elapsed2);

  // Compare results
  for (int i = 0; i < width*width; i++) {
    if (c_h[i] != d_h[i]) {
      printf("Error: CPU and GPU results do not match\n");
      break;
    }
  }

  // Start timer for GPU (optimised)
  hipEventRecord(start, 0);

  // Launch kernel (optimised)
  matrixMultiplyOptimised<<<grid, block>>>(a_h, b_h, c_h, width);

  // Stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed3, start, stop);

  // Print execution time
  printf("Time to calculate results on GPU (optimised): %f ms\n", elapsed3);

  // Copy results to host
  hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

  // Compare results
  for (int i = 0; i < width*width; i++) {
    if (c_h[i] != d_h[i]) {
      printf("Error: CPU and GPU (optimised) results do not match\n");
      break;
    }
  }

  // Free memory
  free(a_h);
  free(b_h);
  free(c_h);
  free(d_h);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
