#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void reduceKernel(int *input, int *output, int N)
{
	int tid = threadIdx.x;
	int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	extern __shared__ int sdata[];
	sdata[tid] = 0;

	//perform first level of reduction, reading from global memory, writing to shared memory
	int sum = (i < N) ? input[i] : 0;

	if (i + blockDim.x < N) sum += input[i+blockDim.x];
	sdata[tid] = sum;

	//synchronise threads in this block before manipulating with the data
	__syncthreads();

	//do reduction in shared memory
	for (int s=blockDim.x/2; s>32; s>>=1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

    if (tid < 32)
    {
        //now that we are using warp-synchronous programming (below) we need to declare our shared memory
        //volatile so that the compiler doesn't reorder stores to it and induce incorrect behavior
        volatile int* smem = sdata;
        if (N >=  64) { smem[tid] += smem[tid + 32]; }
        if (N >=  32) { smem[tid] += smem[tid + 16]; }
        if (N >=  16) { smem[tid] += smem[tid +  8]; }
        if (N >=   8) { smem[tid] += smem[tid +  4]; }
        if (N >=   4) { smem[tid] += smem[tid +  2]; }
        if (N >=   2) { smem[tid] += smem[tid +  1]; }
    }

	//write result for this block to global mem
	if(tid == 0) output[blockIdx.x] = sdata[0];
}

int nextPow2(int x)
{
    --x;

    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;

    return ++x;
}

int main(int argc, char **argv)
{
	//number of elements in the array
	int N = 4000000;

	//set the number of threads
	int maxThreads = 128;

	//grid and block sizes
	int threads = (N < maxThreads*2) ? nextPow2((N + 1)/ 2) : maxThreads;
    int blocks = (N + (threads * 2 - 1)) / (threads * 2);	
	dim3 grid(blocks, 1, 1);
	dim3 block(threads, 1, 1);

	//print the number of elements
	printf("\n======================\n");
	printf("Parallel reduction sum\n");
	printf("======================\n\n");
	printf("Total number of elements to sum: %i\n", N);
	printf("Kernel launch configuration: %i blocks of %i threads\n", grid.x, block.x);

	//host memory pointer
	int *data_h;

	//allocate host memory
	data_h = (int*)malloc(N*sizeof(int));

	//initialise random number generator seed based on current time
	srand(time(NULL));

	//generate data
	for (int i=0; i<N; i++) data_h[i] = 1;

	//device memory pointers
	int *data_d;
	int *blockSum_d;
	
	//allocate device memory
	hipMalloc((void **)&data_d, N * sizeof(int));
	hipMalloc((void **)&blockSum_d, grid.x * sizeof(int));

	//copy memory to device
	hipMemcpy(data_d, data_h, N * sizeof(int), hipMemcpyHostToDevice);

	//calculate sums on device
	float timeGPU;
	hipEvent_t start;     
	hipEvent_t stop;
	hipEventCreate(&start);     		
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	//level 0
	printf("Level 0 kernel summing %i elements with %i blocks of %i threads...\n", N, grid.x, block.x);
	reduceKernel<<<grid, block, block.x*sizeof(int)>>>(data_d, blockSum_d, N);
	//level 1+
	int remainingElements = grid.x;
	int level = 1;
	while(remainingElements > 1)
	{
		threads = (remainingElements < maxThreads*2) ? nextPow2((remainingElements + 1)/ 2) : maxThreads;
		blocks = (remainingElements + (threads * 2 - 1)) / (threads * 2);	

		printf("Level %i kernel summing %i elements with %i blocks of %i threads...\n", level, remainingElements, blocks, threads);
		reduceKernel<<<blocks, threads, threads*sizeof(int)>>>(blockSum_d, blockSum_d, remainingElements);
		remainingElements = blocks;

		level++;
	}
	hipEventRecord(stop, 0);     		
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timeGPU, start, stop);

	//copy results back to host
	int sumGPU;
	hipMemcpy(&sumGPU, blockSum_d, sizeof(int), hipMemcpyDeviceToHost);

	//print result
	printf("result: %i   time: %f ms   throughput: %.4f GB/s\n", sumGPU, timeGPU, 1.0e-9 * ((double)N*sizeof(int))/(timeGPU/1000));

	//hipDeviceReset must be called before exiting in order for profiling and tracing tools such as Nsight and Visual Profiler to show complete traces
    hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	else return 0;
}