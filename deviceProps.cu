#include "hip/hip_runtime.h"

#include <stdio.h>

int main() {
  // Get number of GPUs
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  printf("Number of GPU devices: %i\n", deviceCount);

  // Get CUDA driver and runtime version
  int driverVersion;
  int runtimeVersion;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  printf("CUDA Driver Version / Runtime Version: %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);

  // Get device properties
  hipDeviceProp_t deviceProperties;
  for (int i = 0; i < deviceCount; i++) {
    hipGetDeviceProperties(&deviceProperties, i);
    printf("Name: %s\n", deviceProperties.name);
  }

  return 0;
}
