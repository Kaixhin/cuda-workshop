#include "hip/hip_runtime.h"

#include <stdio.h>
#define N_THREADS 1024

// f(x) = 1 / (1 + e^-x)
__global__ void sigmoidKernel(int *a, int *c, int N) {
  int tdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tdx < N) { // Boundary condition
    c[tdx] = 1 / (1 + __expf(a[tdx]));
  }
}

int main() {
  int N = 4096000; // Array size

  // Host pointers
  int *a_h[2], *b_h[2];
  // Device pointers
  int *a_d[2], *b_d[2];

  hipStream_t stream[2];
  for (int i = 0; i < 2; ++i) {
    hipStreamCreate(&stream[i]); // Stream creation

    // Allocate pinned memory 
    hipHostMalloc((void**)&a_h[i], (N/2)*sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&b_h[i], (N/2)*sizeof(int), hipHostMallocDefault);

    // Allocate device memory
    hipMalloc((void**)&a_d[i], (N/2)*sizeof(int));
    hipMalloc((void**)&b_d[i], (N/2)*sizeof(int));
  }

  // Load (split) input array with numbers
  for (int i = 0; i < 2; i++) {
    for (int j = 0; j < N/2; j++) {
      a_h[i][j] = i * N/2 + j;
    }
  }

  // Create timer
  hipEvent_t start;
  hipEvent_t stop;
  float elapsedTime;
  // Start timer
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Streams
  for (int i = 0; i < 2; i++) {
    dim3 grid(N/2 / N_THREADS, 1, 1);
    dim3 block(N_THREADS, 1, 1);
    hipMemcpyAsync(a_d[i], a_h[i], (N/2)*sizeof(int), hipMemcpyHostToDevice, stream[i]);
    sigmoidKernel<<<grid, block, 0, stream[i]>>>(a_d[0], b_d[0], N);
    hipMemcpyAsync(b_h[i], b_d[i], (N/2)*sizeof(int), hipMemcpyDeviceToHost, stream[i]);
  }

  // Stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  // Print execution time
  printf("Time to calculate results: %f ms\n", elapsedTime);

  // Clean up
  for (int i = 0; i < 2; i++) {
    hipStreamDestroy(stream[i]);
    hipHostFree(a_h[i]);
    hipHostFree(b_h[i]);
  }
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipDeviceReset();

  return 0;
}
