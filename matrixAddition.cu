#include <stdio.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 1024

void matrixAdd(int *a, int *b, int *c, int N) {
  int index;
  for (int col = 0; col < N; col++) {
    for (int row = 0; row < N; row++) {
      c[index] = a[index] + b[index];
    }
  }
}

__global__ void matrixAddKernel(int *a, int *b, int *c, int N) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int index = row * N + col;
  c[index] = a[index] + b[index];
}

int main() {
  int N = 4096; // Define size of 1 side of square matrix
  // Initialise grid and block variables
  dim3 grid(N / THREADS_PER_BLOCK, 1, 1);
  dim3 block(THREADS_PER_BLOCK, 1, 1);

  // Initialise host pointers (dynamically allocated memory) and device pointers
  int *a_h;
  int *b_h;
  int *c_h; // GPU results
  int *d_h; // CPU results
  int *a_d;
  int *b_d;
  int *c_d;

  int size; // Number of bytes required by arrays

  // Create timer
  hipEvent_t start;
  hipEvent_t stop;
  float elapsedTime;

  // Print out information about blocks and threads
  printf("Number of threads: %i (%ix%i)\n", block.x*block.y, block.x, block.y);
  printf("Number of blocks: %i (%ix%i)\n", grid.x*grid.y, grid.x, grid.y);

  // Dynamically allocate host memory
  size = N * N * sizeof(int);
  
  a_h = (int*) malloc(size);
  b_h = (int*) malloc(size);
  c_h = (int*) malloc(size);
  d_h = (int*) malloc(size);

  // Load host arrays with data
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      a_h[i * N + j] = i;
      b_h[i * N + j] = i;
    }
  }

  // Allocate device memory
  hipMalloc((void**)&a_d, size);
  hipMalloc((void**)&b_d, size);
  hipMalloc((void**)&c_d, size);

  // Copy host memory to device memory
  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
  hipMemcpy(c_d, c_h, size, hipMemcpyHostToDevice);

  // Start timer for GPU
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Launch kernel
  matrixAddKernel<<<grid, block>>>(a_d, b_d, c_d, N);

  // Stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  // Print execution time
  printf("Time to calculate results on GPU: %f ms\n", elapsedTime);

  // Copy results to device
  hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
  
  // Start timer for CPU
  hipEventRecord(start, 0);

  // Launch CPU code
  matrixAdd(a_h, b_h, d_h, N);

  // Stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  // Print execution time
  printf("Time to calculate results on CPU: %f ms\n", elapsedTime);

  // Compare results
  for (int i = 0; i < N*N; i++) {
    if (c_h[i] != d_h[i]) {
      printf("Error: CPU and GPU results do not match\n");
      break;
    }
  }

  // Free memory
  free(a_h);
  free(b_h);
  free(c_h);
  free(d_h);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
